#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

#define BLOCK_SIZE 256
#define CUDA_CORES 768

__global__ void vectorAddShared(int* A, int* B, int* C, int n) {
    __shared__ int s_A[BLOCK_SIZE];
    __shared__ int s_B[BLOCK_SIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        s_A[threadIdx.x] = A[idx];
        s_B[threadIdx.x] = B[idx];
        __syncthreads();

        C[idx] = s_A[threadIdx.x] + s_B[threadIdx.x];
    }
}

void vectorAddCPU(int* A, int* B, int* C, int n) {
    for (int i = 0; i < n; ++i)
        C[i] = A[i] + B[i];
}

int main() {
    int sizes[5] = {1000000, 5000000, 8000000, 10000000, 30000000};
    cout << "\nVector Addition Benchmark (Shared Memory)\n";
    cout << "--------------------------------------------------------------------------\n";
    cout << "| " << setw(12) << "Vector Size" 
         << " | " << setw(12) << "CPU Time(s)"
         << " | " << setw(12) << "GPU Time(s)"
         << " | " << setw(8) << "Speedup"
         << " | " << setw(10) << "Efficiency" << " |\n";
    cout << "--------------------------------------------------------------------------\n";

    for (int i = 0; i < 5; i++) {
        int N = sizes[i];
        int* h_A = (int*)malloc(N * sizeof(int));
        int* h_B = (int*)malloc(N * sizeof(int));
        int* h_C_CPU = (int*)malloc(N * sizeof(int));
        int* h_C_GPU = (int*)malloc(N * sizeof(int));

        for (int j = 0; j < N; ++j) {
            h_A[j] = rand() % 100;
            h_B[j] = rand() % 100;
        }

        // CPU time
        auto start_cpu = chrono::high_resolution_clock::now();
        vectorAddCPU(h_A, h_B, h_C_CPU, N);
        auto end_cpu = chrono::high_resolution_clock::now();
        chrono::duration<double> cpu_duration = end_cpu - start_cpu;
        double cpu_time = cpu_duration.count();

        // Allocate device memory
        int *d_A, *d_B, *d_C;
        hipMalloc((void**)&d_A, N * sizeof(int));
        hipMalloc((void**)&d_B, N * sizeof(int));
        hipMalloc((void**)&d_C, N * sizeof(int));

        hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, N * sizeof(int), hipMemcpyHostToDevice);

        // GPU time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int gridSize = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

        hipEventRecord(start);
        vectorAddShared<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float gpu_time_ms;
        hipEventElapsedTime(&gpu_time_ms, start, stop);
        double gpu_time = gpu_time_ms / 1000.0;

        hipMemcpy(h_C_GPU, d_C, N * sizeof(int), hipMemcpyDeviceToHost);

        double speedup = cpu_time / gpu_time;
        double efficiency = speedup / CUDA_CORES;

        cout << "| " << setw(12) << N
             << " | " << setw(12) << fixed << setprecision(6) << cpu_time
             << " | " << setw(12) << fixed << setprecision(6) << gpu_time
             << " | " << setw(8) << fixed << setprecision(2) << speedup
             << " | " << setw(10) << fixed << setprecision(6) << efficiency
             << " |\n";

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(h_A); free(h_B); free(h_C_CPU); free(h_C_GPU);
        hipEventDestroy(start); hipEventDestroy(stop);
    }

    cout << "--------------------------------------------------------------------------\n";
    return 0;
}
